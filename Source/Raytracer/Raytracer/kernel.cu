
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include "Math/vec3.h"
#include "Math/Ray.h"
#include "Raytracer.h"


int main()
{
    Raytracer* rt = new Raytracer("", "C:\\Users\\Student\\OneDrive - Neumont College of Computer Science\\Q9 FALL 2020\\Capstone Project\\CapstoneWork\\Source\\Raytracer\\Outputs\\output.ppm");

    
    rt->StartRender();
    rt->WriteToFile();

    std::cout << "File written!";
    //CheckCudaErrors(hipFree(frameBuffer));
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

