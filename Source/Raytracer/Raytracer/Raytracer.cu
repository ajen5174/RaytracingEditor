#include "hip/hip_runtime.h"
#include "Raytracer.h"
#include "Math/Sphere.h"
#include "Math/Triangle.h"



__global__ void CreateWorld(Hittable** object, Hittable** object2, Camera** cam, int width, int height)
{
    *cam = new Camera(vec3(0.0f), vec3(0.0f), vec3(0.0f), 45.0f, (float)width / (float)height);
    //*object = new Sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f);
    *object = new Triangle(vec3(-0.5f, -0.5f, -1.0f), vec3(0.5f, -0.5f, -1.0f), vec3(0.0f, 0.5f, -1.0f));
    *object2 = new Triangle(vec3(-0.5f, -0.5f, -1.0f), vec3(-1.75f, 0.0f, -3.0f), vec3(0.0f, 0.5f, -1.0f));

}


__device__ vec3 GetColor(Hittable** object, Hittable** object2, const Ray& r)
{
    HitInfo info;
    if ((*object)->Hit(r, 0.0f, 100.0f, info))
    {
        if (info.u >= 0.0f)
        {
            return (info.normal + vec3(1.0f)) / 2.0f;
            return vec3(1.0f * info.u, 1.0f * info.v, 1.0f * info.w);
        }
    }
    else
    {
        if ((*object2)->Hit(r, 0.0f, 100.0f, info))
        {

            if (info.u >= 0.0f)
            {
                return (info.normal + vec3(1.0f)) / 2.0f;
                return vec3(1.0f * info.u, 1.0f * info.v, 1.0f * info.w);
            }
        }
    }
    
    vec3 unitDir = Normalize(r.direction);
    float t = 0.5f * (unitDir.y + 1.0f);//based on how high it is, change the weight of the color from white to light blue
    return (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
}

__global__ void Render(vec3* frameBuffer, int width, int height, Camera** cam, Hittable** object, Hittable** object2) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= width) || (j >= height)) return;
    int pixel_index = j * width + i;
    float u = (float)i / (float)width;
    float v = (float)j / (float)height;
    frameBuffer[pixel_index] = GetColor(object, object2, (*cam)->GetRay(u, v));

}



void CheckCuda(hipError_t result, char const* const func, const char* const file, int const line)
{
if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
}
}

bool Raytracer::LoadScene(std::string sceneToLoad)
{

    return false;
}



Raytracer::Raytracer(std::string sceneToLoad, std::string renderPath)
    :renderPath(renderPath)
{
    LoadScene(sceneToLoad);
}

bool Raytracer::StartRender()
{
    width = 800;
    height = 600;
    int numPixels = width * height;
    size_t frameBufferSize = numPixels * sizeof(vec3);

    CheckCudaErrors(hipMallocManaged((void**)&frameBuffer, frameBufferSize));
    int threadX = 8;
    int threadY = 8;

    clock_t start, stop;
    start = clock();

    
    Camera** cam;
    CheckCudaErrors(hipMalloc((void**)&cam, sizeof(Camera*)));

    Hittable** object;
    CheckCudaErrors(hipMalloc((void**)&object, sizeof(Hittable*)));
    
    Hittable** object2;
    CheckCudaErrors(hipMalloc((void**)&object2, sizeof(Hittable*)));

    CreateWorld<<<1, 1>>>(object, object2, cam, width, height);

    dim3 blocks(width / threadX + 1, height / threadY + 1);
    dim3 threads(threadX, threadY);

    Render <<<blocks, threads>>> (frameBuffer, width, height, cam, object, object2);
    CheckCudaErrors(hipGetLastError());
    CheckCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Render took " << seconds << " seconds.\n";
    return false;
}

void Raytracer::WriteToFile()
{

    //not sure why but it won't use relative paths here?
    std::ofstream myFile(renderPath);

    // Output FB as Image
    myFile << "P3\n" << width << " " << height << "\n255\n";
    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            size_t pixel_index = j * width + i;
            vec3 color = frameBuffer[pixel_index];
            int ir = int(255.99 * color.x);
            int ig = int(255.99 * color.y);
            int ib = int(255.99 * color.z);
            myFile << ir << " " << ig << " " << ib << "\n";
        }
    }
}
